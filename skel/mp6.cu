#include "hip/hip_runtime.h"
// The <expected_output_file> and <input_file_n> are the input and output files provided in the dataset.
// The <output_file> is the location you��d like to place the output from your program.
// The <type> is the output file type: vector, matrix, or image.
// If an MP does not expect an input or output, then pass none as the parameter.

#include    <wb.h>

#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

#define Mask_width  5
#define Mask_radius Mask_width/2

//@@ INSERT CODE HERE

#define TILE_WIDTH 16
#define clamp(x) (min(max((x), 0.0), 1.0))
 
//@@ INSERT CODE HERE
__global__ void convolution(float *I, const float* __restrict__ M, float *P,
                            int channels, int width, int height) {

	__shared__ float Ns[TILE_WIDTH + Mask_width - 1][TILE_WIDTH + Mask_width - 1];
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int row_o = blockIdx.y * TILE_WIDTH + ty;
	int col_o = blockIdx.x * TILE_WIDTH + tx;
	int row_i = row_o - Mask_radius;
	int col_i = col_o - Mask_radius;
	
	int i, j, k;
	for (k = 0; k < channels; k++) {
		if((row_i >= 0) && (row_i < height) && (col_i >= 0)  && (col_i < width) ) {
			Ns[ty][tx] = I[(row_i*width + col_i)*channels + k];
		} else{
			Ns[ty][tx] = 0.0f;
		}
		__syncthreads();
		float accum = 0;
		if(tx < TILE_WIDTH && ty < TILE_WIDTH){
			for (j = 0; j < Mask_width; j++){
				for (i = 0; i < Mask_width; i++){
					accum += Ns[threadIdx.y + j][threadIdx.x + i] * M[j * Mask_width + i];
				}
			}
			if (row_o < height && col_o < width){
				P[(row_o * width + col_o) * channels + k] = clamp(accum);
			}
		}
		__syncthreads();
	}
}

int main(int argc, char* argv[]) {
    wbArg_t args;
    int maskRows;
    int maskColumns;
    int imageChannels;
    int imageWidth;
    int imageHeight;
    char * inputImageFile;
    char * inputMaskFile;
    wbImage_t inputImage;
    wbImage_t outputImage;
    float * hostInputImageData;
    float * hostOutputImageData;
    float * hostMaskData;
    float * deviceInputImageData;
    float * deviceOutputImageData;
    float * deviceMaskData;

    args = wbArg_read(argc, argv); /* parse the input arguments */

    inputImageFile = wbArg_getInputFile(args, 0);
    inputMaskFile = wbArg_getInputFile(args, 1);

    inputImage = wbImport(inputImageFile);
    hostMaskData = (float *) wbImport(inputMaskFile, &maskRows, &maskColumns);

    assert(maskRows == 5); /* mask height is fixed to 5 in this mp */
    assert(maskColumns == 5); /* mask width is fixed to 5 in this mp */

    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);

    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);

    hostInputImageData = wbImage_getData(inputImage);
    hostOutputImageData = wbImage_getData(outputImage);

    wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

    wbTime_start(GPU, "Doing GPU memory allocation");
    hipMalloc((void **) &deviceInputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceOutputImageData, imageWidth * imageHeight * imageChannels * sizeof(float));
    hipMalloc((void **) &deviceMaskData, maskRows * maskColumns * sizeof(float));
    wbTime_stop(GPU, "Doing GPU memory allocation");


    wbTime_start(Copy, "Copying data to the GPU");
    hipMemcpy(deviceInputImageData,
               hostInputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyHostToDevice);
    hipMemcpy(deviceMaskData,
               hostMaskData,
               maskRows * maskColumns * sizeof(float),
               hipMemcpyHostToDevice);
    wbTime_stop(Copy, "Copying data to the GPU");


    wbTime_start(Compute, "Doing the computation on the GPU");
    //@@ INSERT CODE HERE
	dim3 dimGrid(ceil((float)imageWidth/TILE_WIDTH), ceil((float)imageHeight/TILE_WIDTH));
    dim3 dimBlock(TILE_WIDTH + Mask_width, TILE_WIDTH + Mask_width, 1);
    convolution<<<dimGrid, dimBlock>>>(deviceInputImageData, deviceMaskData, deviceOutputImageData,
                                       imageChannels, imageWidth, imageHeight);
	
    wbTime_stop(Compute, "Doing the computation on the GPU");


    wbTime_start(Copy, "Copying data from the GPU");
    hipMemcpy(hostOutputImageData,
               deviceOutputImageData,
               imageWidth * imageHeight * imageChannels * sizeof(float),
               hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying data from the GPU");

    wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

    wbSolution(args, outputImage);

    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceMaskData);

    free(hostMaskData);
    wbImage_delete(outputImage);
    wbImage_delete(inputImage);

    return 0;
}
